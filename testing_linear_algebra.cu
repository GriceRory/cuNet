#include "hip/hip_runtime.h"
#include "linear_algebra.h"


void testMatrixMemoryFunctions();
void testVectorMemoryFunctions();
void testAlgebraFunctions();
void testMatrixMultiply();
void testVectorAdd();
void testMatrixAdd();

void testMatrixMemoryFunctions(){
	int failed = 0;
	printf("testing matrix memory functions\n");
	int height = 56;
	int width = 51;
	float max = 20.0;
	matrix d_A;
	matrix A;
	matrix B;
	cudaBuildMatrix(&d_A, height, width);
	A = buildMatrix(height, width);
	B = buildMatrix(height, width);
	randomizeMatrix(&A, max);

	int i = copyHostToDevice(&A, &d_A);
	int j = copyDeviceToHost(&d_A, &B);

	int k = cudaFreeMatrix(&d_A);

	for(int i = 0; i < height; i++){
		for(int j = 0; j < width; j++){
			if(getElement(A, i, j) != getElement(B, i, j)){
				printf("failed on A[%d][%d]=%f, B[%d][%d]=%f\n", i, j, getElement(A, i, j), i, j, getElement(B, i, j));
				failed = 1;
			}
		}
	}


	if(!failed || i != hipSuccess || j != hipSuccess){
		printf("successfully tested matrix memory functions\n\n\n");
	}else{
		printf("failed, i = %d, j = %d\n", i, j);
		printMatrix(A);
		printf("\n\n");
		printMatrix(B);
	}
}
void testVectorMemoryFunctions(){
	int failed = 0;
	printf("testing vector memory functions\n");
	int length = 21;
	float max = 20.0;
	vector d_A;
	vector A;
	vector B;
	cudaBuildVector(&d_A, length);
	A = buildVector(length);
	B = buildVector(length);
	randomizeVector(A, max);

	int i = copyHostToDevice(&A, &d_A);
	int j = copyDeviceToHost(&d_A, &B);

	cudaFreeVector(&d_A);

	for(int i = 0; i < length; i++){
		if(getElement(A, i) != getElement(B, i)){
			printf("failed on A[%d]=%f, B[%d]=%f\n", i, getElement(A, i), i, getElement(B, i));
			failed = 1;
		}
	}

	if(!failed){printf("successfully tested vector memory functions\n\n\n");
	}else{
		printf("failed\n");
		printVector(A);
		printf("\n\n");
		printVector(B);
	}
}

void testAlgebraFunctions(){
	printf("testing algebra functions\n");

	testMatrixMultiply();
	//testVectorAdd();
	//testMatrixAdd();
}

void testMatrixMultiply(){
	int failed = 0;
	int height = 5;
	int width = 5;
	int length = height;
	float max = 10.0;
	printf("testing matrix multiply\n");

	matrix M;
	matrix d_M;
	vector in, out;
	vector d_in, d_out;
	M = buildMatrix(height, width);
	int build_d_M = cudaBuildMatrix(&d_M, height, width);
	in = buildVector(length);
	out = buildVector(length);
	int build_d_in = cudaBuildVector(&d_in, length);
	int build_d_out = cudaBuildVector(&d_out, length);
	randomizeMatrix(&M, max);
	randomizeVector(in, max);
	int vector_copy_host_to_device = copyHostToDevice(&in, &d_in);
	int matrix_copy_host_to_device = copyHostToDevice(&M, &d_M);
	printf("%d, %d, %d, %d, %d\n", build_d_M, build_d_in, build_d_out, vector_copy_host_to_device, matrix_copy_host_to_device);
	int threads_per_block = BLOCK_SIZE;
	int blocks = width;
	matrixMultiply<<<threads_per_block, blocks>>>(d_in, M, d_out);
	printf("%d\n", hipGetLastError());
	hipDeviceSynchronize();
	printf("%d\n", hipGetLastError());
	int vector_copy_device_to_host = copyDeviceToHost(&d_out, &out);
	printf("%d\n", hipGetLastError());

	printf("in \n");
	printVector(in);
	printf("\nmatrix \n");
	printMatrix(M);
	printf("\nout\n");
	printVector(out);

	for(int i = 0; i < width; i++){
		float temp = 0.0;
		for(int j = 0; j < height; j++){
			temp += getElement(M, i, j) * getElement(in, j);
		}
		if(getElement(out, i) != temp){
			printf("failed on index %d with out = %.3f, expected = %.3f\n", i, getElement(out, i), temp);
		}
	}
	cudaFreeMatrix(&M);
	cudaFreeVector(&d_in);
	cudaFreeVector(&d_out);

	if(failed){printf("successfully tested matrix multiplication\n\n\n");
	}else{printf("failed\n");}
}

void testVectorAdd(){
	int failed = 0;
	printf("testing vector addition\n");
	int length = 20;
	float max = 20.0;
	vector v = buildVector(length);
	vector w = buildVector(length);
	vector u = buildVector(length);
	vector d_v, d_w;
	cudaBuildVector(&d_v, length);
	cudaBuildVector(&d_w, length);

	randomizeVector(v, max);
	randomizeVector(w, max);

	copyHostToDevice(&v, &d_v);
	copyHostToDevice(&w, &d_w);

	int threads_per_block = BLOCK_SIZE;
	int blocks = (length / BLOCK_SIZE) + 1;
	vectorAdd<<<threads_per_block, blocks>>>(d_v, d_w);
	copyHostToDevice(&u, &d_v);

	for(int i = 0; i < length; i++){
		if(getElement(u, i) != getElement(v, i) + getElement(w, i)){
			failed = 1;
			printf("failed on element %d with u%.3f != v%.3f + w%.3f\n", i, getElement(u, i), getElement(v, i), getElement(w, i));
		}
	}

	if(failed){printf("successfully tested vector addition\n\n\n");
	}else{
		printf("failed\n");
	}
}

void testMatrixAdd(){}


int main(){
	srand(time(NULL));
	//testMatrixMemoryFunctions();
	//testVectorMemoryFunctions();
	testAlgebraFunctions();
}
