#include "hip/hip_runtime.h"
#include "linear_algebra.h"


void testMatrixMemoryFunctions();
void testVectorMemoryFunctions();
void testAlgebraFunctions();
void testMatrixMultiply();
void testVectorAdd();
void testMatrixAdd();

void testMatrixMemoryFunctions(){
	int failed = 0;
	printf("testing matrix memory functions\n");
	int height = 56;
	int width = 51;
	float max = 20.0;
	matrix d_A;
	matrix A;
	matrix B;
	cudaBuildMatrix(&d_A, height, width);
	buildMatrix(&A, height, width);
	buildMatrix(&B, height, width);
	randomizeMatrix(&A, max);

	int i = copyHostToDevice(&A, &d_A);
	int j = copyDeviceToHost(&d_A, &B);

	cudaFreeMatrix(&d_A);

	for(int i = 0; i < height; i++){
		for(int j = 0; j < width; j++){
			if(getElement(A, i, j) != getElement(B, i, j)){
				printf("failed on A[%d][%d]=%f, B[%d][%d]=%f\n", i, j, getElement(A, i, j), i, j, getElement(B, i, j));
				failed = 1;
			}
		}
	}


	if(!failed){printf("successfully tested matrix memory functions\n");
	}else{
		printf("failed\n");
		printMatrix(A);
		printf("\n\n");
		printMatrix(B);
	}
}
void testVectorMemoryFunctions(){
	int failed = 0;
	printf("testing vector memory functions\n");
	int length = 21;
	float max = 20.0;
	vector d_A;
	vector A;
	vector B;
	cudaBuildVector(&d_A, length);
	A = buildVector(length);
	B = buildVector(length);
	randomizeVector(A, max);

	int i = copyHostToDevice(&A, &d_A);
	int j = copyDeviceToHost(&d_A, &B);

	cudaFreeVector(&d_A);

	for(int i = 0; i < length; i++){
		if(getElement(A, i) != getElement(B, i)){
			printf("failed on A[%d]=%f, B[%d]=%f\n", i, getElement(A, i), i, getElement(B, i));
			failed = 1;
		}
	}

	if(!failed){printf("successfully tested vector memory functions\n");
	}else{
		printf("failed\n");
		printVector(A);
		printf("\n\n");
		printVector(B);
	}
}

void testAlgebraFunctions(){
	int failed = 0;
	printf("testing algebra functions\n");

	testMatrixMultiply();
	testVectorAdd();
	testMatrixAdd();

	if(!failed){printf("successfully tested algebra functions\n");
	}else{printf("failed\n");}
}

//not finished
void testMatrixMultiply(){
	int failed = 0;
	int height = 5;
	int width = 5;
	int length = height;
	float max = 20.0;
	printf("testing algebra functions\n");

	matrix M;
	matrix d_M;
	vector in, out;
	vector d_in, d_out;
	buildMatrix(&M, height, width);
	in = buildVector(length);
	out = buildVector(length);
	cudaBuildVector(&d_in, length);
	cudaBuildVector(&d_out, length);
	randomizeMatrix(&M, max);
	randomizeVector(in, max);
	copyHostToDevice(&in, &d_in);
	copyHostToDevice(&M, &d_M);

	int threads_per_block = BLOCK_SIZE;
	int blocks = width;
	matrixMultiply<<<threads_per_block, blocks>>>(d_in, M, d_out);
	copyDeviceToHost(&d_out, &out);

	printf("in \n");
	printVector(in);
	printf("\nmatrix \n");
	printMatrix(M);
	printf("\nout\n");
	printVector(out);

	for(int i = 0; i < width; i++){
		float temp = 0.0;
		for(int j = 0; j < height; j++){
			temp += getElement(M, i, j) * getElement(in, j);
		}
		if(getElement(out, i) != temp){
			printf("failed on index %d with out = %.3f, expected = %.3f\n", i, getElement(out, i), temp);
		}
	}
	cudaFreeMatrix(&M);
	cudaFreeVector(&d_in);
	cudaFreeVector(&d_out);

	if(!failed){printf("successfully tested matrix multiplication\n");
	}else{printf("failed\n");}
}

void testVectorAdd(){
	int failed = 0;
	printf("testing algebra functions\n");
	int length = 20;
	float max = 20.0;
	vector v = buildVector(length);
	vector w = buildVector(length);
	vector u = buildVector(length);
	vector d_v, d_w;
	cudaBuildVector(&d_v, length);
	cudaBuildVector(&d_w, length);

	randomizeVector(v, max);
	randomizeVector(w, max);

	copyHostToDevice(&v, &d_v);
	copyHostToDevice(&w, &d_w);

	int threads_per_block = BLOCK_SIZE;
	int blocks = (length / BLOCK_SIZE) + 1;
	vectorAdd<<<threads_per_block, blocks>>>(d_v, d_w);
	copyHostToDevice(&u, &d_v);

	for(int i = 0; i < length; i++){
		if(getElement(u, i) != getElement(v, i) + getElement(w, i)){
			failed = 1;
			printf("failed on element %d with u%.3f != v%.3f + w%.3f\n", i, getElement(u, i), getElement(v, i), getElement(w, i));
		}
	}

	if(!failed){printf("successfully tested algebra functions\n");
	}else{
		printf("failed\n");
	}
}

void testMatrixAdd(){}


int main(){
	srand(time(NULL));
	testMatrixMemoryFunctions();
	testVectorMemoryFunctions();
	testAlgebraFunctions();
}
