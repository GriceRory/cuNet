int difference_tollerance(float actual, float expected, float tollerance){
	return actual-expected > tollerance || expected-actual > tollerance;
}

#include <time.h>
#include <unistd.h>
hipStream_t *streams;

#include "backpropogation.h"
#include "globals.h"



#include "testing_linear_algebra.h"
#include "testing_network.h"
#include "test_database.h"
#include "testing_backpropogation.h"
#include "test_minst.h"

int main(void){
	srand(time(NULL));
	int fails = 0;
	number_of_streams = 5;
	streams = (hipStream_t*)malloc(sizeof(hipStream_t)*number_of_streams);
	for(int i = 0; i < number_of_streams; ++i){
		hipStreamCreate(&streams[i]);
	}
/*
	fails = test_linear_algebra();
	if(!fails){//returns a failure failure of 1, success of 0
		printf("testing linear_algebra.h nominal\n\n\n");
	}

	int network_fails = test_network();
	fails |= network_fails;
	if(!network_fails){
		printf("testing network.h nominal\n\n\n");
	}

	int database_fails = testing_database();
	fails |= database_fails;
	if(!database_fails){printf("testing database.h nominal\n\n\n");}


	int backpropogation_fails = test_backpropogation();
	fails |= backpropogation_fails;
	if(!backpropogation_fails){printf("testing backpropogation.h nominal\n\n\n");}
*/

	int minst_fails = test_minst();
	fails |= minst_fails;
	if(!minst_fails){printf("test_minst.h nominal\n");}


	if(!fails){
		printf("\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\nall systems nominal\n\n");
	}else{
		printf("\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\n\nsystems failure\n\n");
	}
	return fails;
}
